#include "hip/hip_runtime.h"
#include "bvh.h"
#include "watch.h"
#include <iostream>
#include <fstream>
#include <algorithm>
#include <bitset>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>

#include "Utilities.h"

using namespace std;
extern inline void copyFromCPUtoGPU(void** dst, void* src, int size);
extern inline void copyFromGPUtoCPU(void** dst, void* src, int size);

// Expands a 10-bit integer into 30 bits
// by inserting 2 zeros after each bit.
__device__ unsigned int d_expandBits(unsigned int v)
{
	v = (v * 0x00010001u) & 0xFF0000FFu;
	v = (v * 0x00000101u) & 0x0F00F00Fu;
	v = (v * 0x00000011u) & 0xC30C30C3u;
	v = (v * 0x00000005u) & 0x49249249u;
	return v;
}

// Calculates a 30-bit Morton code for the
// given 3D point located within the unit cube [0,1].
__device__ unsigned int d_morton3D(glm::vec3 p)
{
	float x = p.x, float y = p.y, float z = p.z;
	x = min(max(x * 1024.0f, 0.0f), 1023.0f);
	y = min(max(y * 1024.0f, 0.0f), 1023.0f);
	z = min(max(z * 1024.0f, 0.0f), 1023.0f);
	unsigned int xx = d_expandBits((unsigned int)x);
	unsigned int yy = d_expandBits((unsigned int)y);
	unsigned int zz = d_expandBits((unsigned int)z);
	return xx * 4 + yy * 2 + zz;
}


__global__ void get_bb(int num, int m, Primitive* d_primitives, BBox* d_bb)
{
	unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= num + 1)
		return;
	int div = m / num;
	int res = m % num;
	if (index == num + 1)
	{
		BBox tem_bbox;
		for (int i = m - res; i < m; i++)
		{
			tem_bbox.expand(d_primitives[i].d_get_expand_bbox());
		}
		d_bb[index] = tem_bbox;
	}
	else
	{
		BBox tem_bbox;
		for (int i = 0; i < div; i++)  //use shared to replace
		{
			tem_bbox.expand(d_primitives[i * num + index].d_get_expand_bbox());
		}
		d_bb[index].expand(tem_bbox);
	}
}

__global__ void compute_morton_bbox(int num, Primitive* d_primitives, BBox bb, MortonCode* mortons, BBox* bboxes)
{
	unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= num)
		return;
	BBox tem_bbox = d_primitives[index].d_get_expand_bbox();
	bboxes[index] = tem_bbox;
	mortons[index] = d_morton3D(bb.getUnitcubePosOf(tem_bbox.centroid()));
}

// Expands a 10-bit integer into 30 bits
// by inserting 2 zeros after each bit.
unsigned int BVHAccel::expandBits(unsigned int v)
{
	v = (v * 0x00010001u) & 0xFF0000FFu;
	v = (v * 0x00000101u) & 0x0F00F00Fu;
	v = (v * 0x00000011u) & 0xC30C30C3u;
	v = (v * 0x00000005u) & 0x49249249u;
	return v;
}

// Calculates a 30-bit Morton code for the
// given 3D point located within the unit cube [0,1].
unsigned int BVHAccel::morton3D(float x, float y, float z)
{
	x = min(max(x * 1024.0f, 0.0f), 1023.0f);
	y = min(max(y * 1024.0f, 0.0f), 1023.0f);
	z = min(max(z * 1024.0f, 0.0f), 1023.0f);
	unsigned int xx = expandBits((unsigned int)x);
	unsigned int yy = expandBits((unsigned int)y);
	unsigned int zz = expandBits((unsigned int)z);
	return xx * 4 + yy * 2 + zz;
}

/**
* a wrapper to calculate morton code from
* the position of an object inside the
* unit cube.
*/
unsigned int BVHAccel::morton3D(glm::vec3 pos)
{
	return morton3D(pos.x, pos.y, pos.z);
}

/**
* comparer used to sort primitives acoording
* to their morton code.
*/


BBox BVHAccel::computet_root_bbox(Primitive* d_tem_primitives)
{
	const unsigned int num_threads = 128;
	vector<BBox> c_bb(num_threads + 1);
	BBox* d_bb;

	copyFromCPUtoGPU((void**)&d_bb, &c_bb[0], sizeof(BBox) * c_bb.size());
	get_bb << <1, c_bb.size() >> > (num_threads, _primitives.size(), d_tem_primitives, d_bb);

	BBox* cc_bb, bb;
	copyFromGPUtoCPU((void**)&cc_bb, d_bb, sizeof(BBox) * c_bb.size());
	for (int i = 0; i < c_bb.size(); i++)
	{
		bb.expand(cc_bb[i]);
	}

	hipFree(d_bb);

	return bb;
}

void BVHAccel::compute_bbox_and_morton()
{
	Primitive* d_tem_primitives;
	MortonCode* d_tem_morton_codes;
	BBox* d_tem_bboxes;
	_morton_codes.resize(_primitives.size());
	_bboxes.resize(_primitives.size());

	copyFromCPUtoGPU((void**)&d_tem_primitives, &_primitives[0], sizeof(Primitive) * _primitives.size());
	copyFromCPUtoGPU((void**)&d_tem_morton_codes, &_morton_codes[0], sizeof(MortonCode) * _morton_codes.size());
	copyFromCPUtoGPU((void**)&d_tem_bboxes, &_bboxes[0], sizeof(BBox) * _bboxes.size());

	BBox bb = computet_root_bbox(d_tem_primitives);

	unsigned int numThreads, numBlocks;
	unsigned int blockSize = 512;
	unsigned int n = _primitives.size();
	numThreads = min(blockSize, n);
	numBlocks = (n % numThreads != 0) ? (n / numThreads + 1) : (n / numThreads);

	compute_morton_bbox << <numBlocks, numThreads >> > (n, d_tem_primitives, bb, d_tem_morton_codes, d_tem_bboxes);

	hipMemcpy(&_morton_codes[0], d_tem_morton_codes, sizeof(MortonCode) * _morton_codes.size(), hipMemcpyDeviceToHost);
	hipMemcpy(&_bboxes[0], d_tem_bboxes, sizeof(BBox) * _bboxes.size(), hipMemcpyDeviceToHost);

	hipFree(d_tem_primitives);
	hipFree(d_tem_morton_codes);
	hipFree(d_tem_bboxes);
}

__global__ void init_nodes(BRTreeNode* _nodes, const unsigned int num)
{
	unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= num)
		return;

	BRTreeNode node;
	node.setIdx(index);
	node.bbox = BBox();

	_nodes[index] = node;
}

void BVHAccel::init()
{
	d_bvh = new D_BVH();

	auto size = _sorted_primitives.size();
	numInternalNode = size - 1;
	numLeafNode = size;

	//whether to set h_vertices = NULL before send to gpu?
	copyFromCPUtoGPU((void**)&d_bvh->d_primitives, &_sorted_primitives[0], sizeof(Primitive) * _sorted_primitives.size());
	copyFromCPUtoGPU((void**)&d_sorted_morton_code, &_sorted_morton_codes[0], sizeof(MortonCode) * _sorted_morton_codes.size());
	copyFromCPUtoGPU((void**)&d_bboxes, &_sorted_bboxes[0], sizeof(BBox) * _sorted_bboxes.size());

	//initialize d_leaf_nodes and d_internal_nodes: with a parallel way? ?????
	hipMalloc((void**)&d_bvh->d_leaf_nodes, numLeafNode * sizeof(BRTreeNode));
	hipMalloc((void**)&d_bvh->d_internal_nodes, numInternalNode * sizeof(BRTreeNode));

	int threadPerBlock = DEFAULT_THREAD_PER_BLOCK;
	int numBlock = (numLeafNode + DEFAULT_THREAD_PER_BLOCK - 1) / threadPerBlock;
	init_nodes << <numBlock, threadPerBlock >> > (d_bvh->d_leaf_nodes, numLeafNode);


	threadPerBlock = DEFAULT_THREAD_PER_BLOCK;
	numBlock = (numInternalNode + DEFAULT_THREAD_PER_BLOCK - 1) / threadPerBlock;
	init_nodes << <numBlock, threadPerBlock >> > (d_bvh->d_internal_nodes, numInternalNode);
}

void BVHAccel::build()
{
	//build the bvh
	int threadPerBlock = DEFAULT_THREAD_PER_BLOCK;
	int numBlock = (numInternalNode + DEFAULT_THREAD_PER_BLOCK - 1) / threadPerBlock;
	processInternalNode << <numBlock, threadPerBlock >> > (d_sorted_morton_code, numInternalNode,
		d_bvh->d_leaf_nodes, d_bvh->d_internal_nodes);

	//calculate bounding box
	threadPerBlock = DEFAULT_THREAD_PER_BLOCK;
	numBlock = (numLeafNode + DEFAULT_THREAD_PER_BLOCK - 1) / threadPerBlock;
	calculateBoudingBox << <numBlock, threadPerBlock >> > (d_bboxes, numLeafNode,
		d_bvh->d_leaf_nodes, d_bvh->d_internal_nodes);
}

void BVHAccel::init_primitives(Mesh& body)
{
	//prepare primitives
	obj_vertices.resize(body.vertices.size());
	for (int i = 0; i < body.vertices.size(); i++)
	{
		obj_vertices[i] = glm::vec3(body.vertices[i]);
	}

	safe_cuda(hipMalloc((void**)&d_obj_vertices, sizeof(glm::vec3) * obj_vertices.size()));
	safe_cuda(hipMemcpy(d_obj_vertices, &obj_vertices[0], sizeof(glm::vec3) * obj_vertices.size(), hipMemcpyHostToDevice));

	//create primitives
	glm::vec3* h_obj_vertices = &obj_vertices[0];
	_primitives.resize(body.vertex_indices.size() / 3);

	for (int i = 0; i < _primitives.size(); i++)
	{
		Primitive tem_pri(h_obj_vertices, d_obj_vertices, body.vertex_indices[i * 3 + 0],
			body.vertex_indices[i * 3 + 1],
			body.vertex_indices[i * 3 + 2]);
		_primitives[i] = tem_pri;
	}
}

BVHAccel::BVHAccel(Mesh& body, size_t max_leaf_size) :

	d_bboxes(nullptr),
#ifdef _DEBUG
	h_leaf_nodes(nullptr),
	h_internal_nodes(nullptr),
#endif
	d_sorted_morton_code(nullptr)
{
	init_primitives(body);

	// edge case
	if (_primitives.empty()) {
		return;
	}

	compute_bbox_and_morton();


	// remove duplicates
	vector<unsigned int> indices;
	indices_sort(_morton_codes, indices);
	remove_redundant(_morton_codes, indices);

	filter(_morton_codes, indices, _sorted_morton_codes);
	filter(_primitives, indices, _sorted_primitives);
	filter(_bboxes, indices, _sorted_bboxes);

	// init	GPU data, including d_bboxes,d_primitives, d_sorted_morton_code,d_leaf_nodes, d_internal_nodes 
	init();

	// build the brt tree
	build();
}

BVHAccel::~BVHAccel()
{
	hipFree(d_bboxes);
	hipFree(d_sorted_morton_code);
	hipFree(d_obj_vertices);

	// Free d_bvh here cause it has pointer points to gpu memory
	// and we need to pass the value several times and make sure the 
	// resource not freed, so we can't free it in its own destructor. 
	// \BVHAccel controls the lifetieme of \d_bvh, if the destructor 
	// of \BVHAccel called, which means we can free all the resources in 
	// gpu and cpu(Obviously, this violates the "new" and "free" pair priciple)

	d_bvh->free_memory();
}
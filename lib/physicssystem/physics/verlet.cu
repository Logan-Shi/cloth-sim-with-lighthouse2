#include "hip/hip_runtime.h"
﻿#include "spring.h"
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include <>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

//physics parameter
__constant__ double spring_structure = 1000.0;
__constant__ double spring_bend = 2.0;
__constant__ float damp = -0.02f;
__constant__ float mass = 0.3;
__constant__ float dt = 1 / 100.0f;

__constant__ float gravit_x = 0.0f;   // in y dir
__constant__ float gravit_y = -0.00981f;   // in y dir
__constant__ float gravit_z = 0.0f;   // in y dir
__constant__ int perm[256] = { 151,160,137,91,90,15,
 131,13,201,95,96,53,194,233,7,225,140,36,103,30,69,142,8,99,37,240,21,10,23,
 190, 6,148,247,120,234,75,0,26,197,62,94,252,219,203,117,35,11,32,57,177,33,
 88,237,149,56,87,174,20,125,136,171,168, 68,175,74,165,71,134,139,48,27,166,
 77,146,158,231,83,111,229,122,60,211,133,230,220,105,92,41,55,46,245,40,244,
 102,143,54, 65,25,63,161, 1,216,80,73,209,76,132,187,208, 89,18,169,200,196,
 135,130,116,188,159,86,164,100,109,198,173,186, 3,64,52,217,226,250,124,123,
 5,202,38,147,118,126,255,82,85,212,207,206,59,227,47,16,58,17,182,189,28,42,
 223,183,170,213,119,248,152, 2,44,154,163, 70,221,153,101,155,167, 43,172,9,
 129,22,39,253, 19,98,108,110,79,113,224,232,178,185, 112,104,218,246,97,228,
 251,34,242,193,238,210,144,12,191,179,162,241, 81,51,145,235,249,14,239,107,
 49,192,214, 31,181,199,106,157,184, 84,204,176,115,121,50,45,127, 4,150,254,
 138,236,205,93,222,114,67,29,24,72,243,141,128,195,78,66,215,61,156,180 }; // perlin noise index & slope

// random parameters
__constant__ float alpha_wind = 0.5;
__constant__ float beta_wind = 0.5;
__constant__ float lambda_wind = 0;
__constant__ float split_time_wind = 300000.0;
// direction parameters
__constant__ float thetav_wind = 90.0;
__constant__ float thetal_wind = 90.0;
__constant__ float degree_to_rad = 3.1415926 / 180.0;
// air parameters
__constant__ float C_lift = 5;
__constant__ float C_drag = 5;
__constant__ float strong_wind = 0.1;

__constant__ float critical_length = 0.03;
__constant__ float repul_stiffness = 200;

__constant__ float slope_index = 1000;
__constant__ float reflect_index = 0.1;
__constant__ float cut_time = 0;


__device__ float perlin(float x) {
	// ÕûÊýx1ºÍx2µÄ×ø±ê
	int x1 = floor(x) + 1;
	int x2 = x1 + 1;

	// x1ºÍx2µÄÌÝ¶ÈÖµ
	float grad1 = perm[x1 % 255] * 2.0 - 255.0;
	float grad2 = perm[x2 % 255] * 2.0 - 255.0;
	// x1ºÍx2Ö¸ÏòxµÄ·½ÏòÏòÁ¿
	float vec1 = x - x1;
	float vec2 = x - x2;

	// xµ½x1µÄ¾àÀë¼´vec1£¬ÀûÓÃ¹«Ê½3¼ÆËãÆ½»¬²ÎÊý
	float t = 3 * pow(vec1, 2) - 2 * pow(vec1, 3);

	// ÌÝ¶ÈÖµÓë·½ÏòÏòÁ¿µÄ³Ë»ý
	float product1 = grad1 * vec1;
	float product2 = grad2 * vec2;

	// ²åÖµ
	return product1 + t * (product2 - product1);
}
__device__ glm::vec3 perlin_noise(glm::vec3 point)
{
	glm::vec3 output_noise;
	float x, y, z;
	float noise_length;
	x = perlin(abs(point.x) * 10000 + 0.1);
	y = perlin(abs(point.y) * 10000 + 0.1);
	z = perlin(abs(point.z) * 10000 + 0.1);
	noise_length = sqrt(pow(x, 2) + pow(y, 2) + pow(z, 2)) + 0.1;
	x = x / noise_length;
	y = y / noise_length;
	z = z / noise_length;
	output_noise = glm::vec3(x, y, z);

	return output_noise;
}
__device__ glm::vec3 compute_wind_force_lift(unsigned int idx, glm::vec3* g_pos_in, unsigned int* CSR_R, s_spring* CSR_C_SPRING, glm::vec3 vel, glm::vec3 pos, float sim_time)
{
	float now_wind = (float)strong_wind * sin(sim_time / 10.0) + 0.2;
	glm::vec3 vec_wind = now_wind * glm::vec3(cos(thetav_wind * degree_to_rad) * sin(thetal_wind * degree_to_rad), cos(thetav_wind * degree_to_rad) * cos(thetal_wind * degree_to_rad), sin(thetav_wind * degree_to_rad));
	glm::vec3 relative_wind = vec_wind - vel;
	int first_neigh = CSR_R[idx];
	glm::vec3 final_normal = glm::vec3(0.0);
	glm::vec3 F_lift = glm::vec3(0.0);
	glm::vec3 F_drag = glm::vec3(0.0);
	float final_S = 0;
	float gap = 0.0001;
	for (int k = first_neigh; k < CSR_R[idx + 1]; k++)
	{
		if (k <= CSR_R[idx + 1] - 2)
		{
			int index_neigh1 = CSR_C_SPRING[k].end;
			int index_neigh2 = CSR_C_SPRING[k + 1].end;
			volatile auto pos1 = g_pos_in[index_neigh1];
			volatile auto pos2 = g_pos_in[index_neigh2];
			glm::vec3 p1 = glm::vec3(pos1.x, pos1.y, pos1.z);
			glm::vec3 p2 = glm::vec3(pos2.x, pos2.y, pos2.z);
			glm::vec3 temp_normal = glm::cross(p1 - pos, p2 - pos);
			float temp_S = 0.5 * glm::length(temp_normal);
			final_normal += temp_normal;
			final_S += temp_S;
		}
		else
		{
			int index_neigh1 = CSR_C_SPRING[k].end;
			int index_neigh2 = CSR_C_SPRING[first_neigh].end;
			volatile auto pos1 = g_pos_in[index_neigh1];
			volatile auto pos2 = g_pos_in[index_neigh2];
			glm::vec3 p1 = glm::vec3(pos1.x, pos1.y, pos1.z);
			glm::vec3 p2 = glm::vec3(pos2.x, pos2.y, pos2.z);
			glm::vec3 temp_normal = glm::cross(p1 - pos, p2 - pos);
			float temp_S = 0.5 * glm::length(temp_normal);
			final_normal += temp_normal;
			final_S += temp_S;
		}
	}
	final_normal = final_normal / (glm::length(final_normal) + gap);
	float costheta = glm::dot(final_normal, glm::normalize(relative_wind));
	F_drag = -C_drag * final_S * sqrt(1 - costheta * costheta) * relative_wind;
	F_lift = C_lift * final_S * costheta * glm::cross(glm::cross(final_normal, relative_wind), relative_wind);

	return F_lift;
}
__device__ glm::vec3 compute_wind_force_drag(unsigned int idx, glm::vec3* g_pos_in, unsigned int* CSR_R, s_spring* CSR_C_SPRING, glm::vec3 vel, glm::vec3 pos, float sim_time)
{
	float now_wind = (float)strong_wind * sin(sim_time / 10.0) + 0.2;
	glm::vec3 vec_wind = now_wind * glm::vec3(cos(thetav_wind * degree_to_rad) * sin(thetal_wind * degree_to_rad), cos(thetav_wind * degree_to_rad) * cos(thetal_wind * degree_to_rad), sin(thetav_wind * degree_to_rad));
	glm::vec3 relative_wind = vec_wind - vel;
	int first_neigh = CSR_R[idx];
	glm::vec3 final_normal = glm::vec3(0.0);
	glm::vec3 F_lift = glm::vec3(0.0);
	glm::vec3 F_drag = glm::vec3(0.0);
	float final_S = 0;
	float gap = 0.0001;
	for (int k = first_neigh; k < CSR_R[idx + 1]; k++)
	{
		if (k <= CSR_R[idx + 1] - 2)
		{
			int index_neigh1 = CSR_C_SPRING[k].end;
			int index_neigh2 = CSR_C_SPRING[k + 1].end;
			volatile auto pos1 = g_pos_in[index_neigh1];
			volatile auto pos2 = g_pos_in[index_neigh2];
			glm::vec3 p1 = glm::vec3(pos1.x, pos1.y, pos1.z);
			glm::vec3 p2 = glm::vec3(pos2.x, pos2.y, pos2.z);
			glm::vec3 temp_normal = glm::cross(p1 - pos, p2 - pos);
			float temp_S = 0.5 * glm::length(temp_normal);
			final_normal += temp_normal;
			final_S += temp_S;
		}
		else
		{
			int index_neigh1 = CSR_C_SPRING[k].end;
			int index_neigh2 = CSR_C_SPRING[first_neigh].end;
			volatile auto pos1 = g_pos_in[index_neigh1];
			volatile auto pos2 = g_pos_in[index_neigh2];
			glm::vec3 p1 = glm::vec3(pos1.x, pos1.y, pos1.z);
			glm::vec3 p2 = glm::vec3(pos2.x, pos2.y, pos2.z);
			glm::vec3 temp_normal = glm::cross(p1 - pos, p2 - pos);
			float temp_S = 0.5 * glm::length(temp_normal);
			final_normal += temp_normal;
			final_S += temp_S;
		}
	}
	final_normal = final_normal / (glm::length(final_normal) + gap);
	float costheta = glm::dot(final_normal, glm::normalize(relative_wind));
	F_drag = -C_drag * final_S * sqrt(1 - costheta * costheta) * relative_wind;
	F_lift = C_lift * final_S * costheta * glm::cross(glm::cross(final_normal, relative_wind), relative_wind);

	return F_drag;
}
__device__ glm::vec3 compute_wind_force2_lift(unsigned int idx, glm::vec3* g_pos_in, unsigned int* CSR_R, s_spring* CSR_C_SPRING, glm::vec3 vel, glm::vec3 pos, glm::vec3 vec_wind)
{
	glm::vec3 relative_wind = vec_wind - vel;
	int first_neigh = CSR_R[idx];
	glm::vec3 final_normal = glm::vec3(0.0);
	glm::vec3 F_lift = glm::vec3(0.0);
	glm::vec3 F_drag = glm::vec3(0.0);
	float final_S = 0;
	float gap = 0.0001;
	for (int k = first_neigh; k < CSR_R[idx + 1]; k++)
	{
		if (k <= CSR_R[idx + 1] - 2)
		{
			int index_neigh1 = CSR_C_SPRING[k].end;
			int index_neigh2 = CSR_C_SPRING[k + 1].end;
			volatile auto pos1 = g_pos_in[index_neigh1];
			volatile auto pos2 = g_pos_in[index_neigh2];
			glm::vec3 p1 = glm::vec3(pos1.x, pos1.y, pos1.z);
			glm::vec3 p2 = glm::vec3(pos2.x, pos2.y, pos2.z);
			glm::vec3 temp_normal = glm::cross(p1 - pos, p2 - pos);
			float temp_S = 0.5 * glm::length(temp_normal);
			final_normal += temp_normal;
			final_S += temp_S;
		}
		else
		{
			int index_neigh1 = CSR_C_SPRING[k].end;
			int index_neigh2 = CSR_C_SPRING[first_neigh].end;
			volatile auto pos1 = g_pos_in[index_neigh1];
			volatile auto pos2 = g_pos_in[index_neigh2];
			glm::vec3 p1 = glm::vec3(pos1.x, pos1.y, pos1.z);
			glm::vec3 p2 = glm::vec3(pos2.x, pos2.y, pos2.z);
			glm::vec3 temp_normal = glm::cross(p1 - pos, p2 - pos);
			float temp_S = 0.5 * glm::length(temp_normal);
			final_normal += temp_normal;
			final_S += temp_S;
		}
	}
	final_normal = final_normal / (glm::length(final_normal) + gap);
	float costheta = glm::dot(final_normal, glm::normalize(relative_wind));
	F_drag = -C_drag * final_S * sqrt(1 - costheta * costheta) * relative_wind;
	F_lift = C_lift * final_S * costheta * glm::cross(glm::cross(final_normal, relative_wind), relative_wind);

	return F_lift;
}
__device__ glm::vec3 compute_wind_force2_drag(unsigned int idx, glm::vec3* g_pos_in, unsigned int* CSR_R, s_spring* CSR_C_SPRING, glm::vec3 vel, glm::vec3 pos, glm::vec3 vec_wind)
{
	glm::vec3 relative_wind = vec_wind - vel;
	int first_neigh = CSR_R[idx];
	glm::vec3 final_normal = glm::vec3(0.0);
	glm::vec3 F_lift = glm::vec3(0.0);
	glm::vec3 F_drag = glm::vec3(0.0);
	float final_S = 0;
	float gap = 0.0001;
	for (int k = first_neigh; k < CSR_R[idx + 1]; k++)
	{
		if (k <= CSR_R[idx + 1] - 2)
		{
			int index_neigh1 = CSR_C_SPRING[k].end;
			int index_neigh2 = CSR_C_SPRING[k + 1].end;
			volatile auto pos1 = g_pos_in[index_neigh1];
			volatile auto pos2 = g_pos_in[index_neigh2];
			glm::vec3 p1 = glm::vec3(pos1.x, pos1.y, pos1.z);
			glm::vec3 p2 = glm::vec3(pos2.x, pos2.y, pos2.z);
			glm::vec3 temp_normal = glm::cross(p1 - pos, p2 - pos);
			float temp_S = 0.5 * glm::length(temp_normal);
			final_normal += temp_normal;
			final_S += temp_S;
		}
		else
		{
			int index_neigh1 = CSR_C_SPRING[k].end;
			int index_neigh2 = CSR_C_SPRING[first_neigh].end;
			volatile auto pos1 = g_pos_in[index_neigh1];
			volatile auto pos2 = g_pos_in[index_neigh2];
			glm::vec3 p1 = glm::vec3(pos1.x, pos1.y, pos1.z);
			glm::vec3 p2 = glm::vec3(pos2.x, pos2.y, pos2.z);
			glm::vec3 temp_normal = glm::cross(p1 - pos, p2 - pos);
			float temp_S = 0.5 * glm::length(temp_normal);
			final_normal += temp_normal;
			final_S += temp_S;
		}
	}
	final_normal = final_normal / (glm::length(final_normal) + gap);
	float costheta = glm::dot(final_normal, glm::normalize(relative_wind));
	F_drag = -C_drag * final_S * sqrt(1 - costheta * costheta) * relative_wind;
	F_lift = C_lift * final_S * costheta * glm::cross(glm::cross(final_normal, relative_wind), relative_wind);

	return F_drag;
}
__device__ glm::vec3 wind_velocity(glm::vec3 F_lift, glm::vec3 F_drag, float sim_time)
{
	glm::vec3 vec_wind = strong_wind * glm::vec3(cos(thetav_wind * degree_to_rad) * sin(thetal_wind * degree_to_rad), cos(thetav_wind * degree_to_rad) * cos(thetal_wind * degree_to_rad), sin(thetav_wind * degree_to_rad));
	vec_wind += alpha_wind * perlin_noise(F_lift) + beta_wind * perlin_noise(F_drag);
	/*
	if (sim_time <= split_time_wind)
	{
		vec_wind += alpha_wind * sim_time * perlin_noise(F_lift) + beta_wind * sim_time * perlin_noise(F_drag);
	}
	else
	{
		glm::vec3 sum_force = alpha_wind * F_lift + beta_wind * F_drag;
		float cx = cos(sum_force.x);
		float cy = cos(sum_force.y);
		float cz = cos(sum_force.z);
		vec_wind += lambda_wind * glm::vec3(cx, cy, cz);
	}


	return vec_wind;
	*/
	return vec_wind;
}
__device__ glm::vec3 compute_selfcollide_force(unsigned int idx, glm::vec3* g_pos_in, unsigned int num, unsigned int* CSR_R1, s_spring* CSR_C_SPRING1, unsigned int* CSR_R2, s_spring* CSR_C_SPRING2)
{
	float dis = 1;
	int first_neigh1 = CSR_R1[idx];
	int last_neigh1 = CSR_R1[idx + 1];
	int first_neigh2 = CSR_R2[idx];
	int last_neigh2 = CSR_R2[idx + 1];
	float temp_stiffness = repul_stiffness;
	float all_neigh[15] = {0};
	for (int i = first_neigh1; i < last_neigh1; i++)
	{
		//all_neigh.push_back(CSR_C_SPRING1[i].end);
		all_neigh[i - first_neigh1] = CSR_C_SPRING1[i].end;
	}
	for (int i = first_neigh2; i < last_neigh2; i++)
	{
		//all_neigh.push_back(CSR_C_SPRING2[i].end);
		all_neigh[last_neigh1 - first_neigh1 + i - first_neigh2] = CSR_C_SPRING2[i].end;
	}
	
	glm::vec3 repul_force = glm::vec3(0.0);
	glm::vec3 pos2;
	volatile glm::vec3 posData = g_pos_in[idx];
	glm::vec3 pos = glm::vec3(posData.x, posData.y, posData.z);
	for (int i = 0; i < num; i++)
	{
		for (int k=0; k < last_neigh2-first_neigh2+last_neigh1-first_neigh1; k++)
		{
			if (all_neigh[k] == i)
			{
				temp_stiffness = 0;
				break;
			}
		}
		volatile glm::vec3 posData = g_pos_in[i];
		pos2 = glm::vec3(posData.x, posData.y, posData.z);
		pos2 = pos2 - pos;
		dis = glm::length(pos2);
		if (dis > 0 && dis < critical_length)
		{
			repul_force -= temp_stiffness * (critical_length - dis) * glm::normalize(pos2);
		}
		temp_stiffness = repul_stiffness;

	}

	return repul_force;
}

__device__ void collision_response_projection(D_BVH bvh,
	glm::vec3& force, glm::vec3& pos, glm::vec3& pos_old,
	int idx, glm::vec3* collision_force, glm::vec3& vel, float sim_time)
{
	glm::vec3 pos1;
	glm::vec3 pos2;
	glm::vec3 dir;
	int idx_pri;
	bool inter = bvh.intersect(pos, idx_pri);
	if (inter)
	{
		float dist;
		float d, d1, d2;
		float mid;
		float gap = 0.00001;
		glm::vec3 normal;
		if (bvh.primitive_intersect(idx_pri, pos, dist, normal))  // check the point inside the primitive or not
		{
			float k = 1.0;
			dist = k * glm::abs(dist);    // //collision response with penalty force

			pos1 = pos + dist * normal;
			d = glm::dot(normal, pos1);
			d2 = glm::dot(normal, pos) - d;
			d1 = glm::dot(normal, pos_old) - d;
			mid = (abs(d2) + 0.00001) / (abs(d1) + abs(d2) + 0.00001);
			pos2 = mid * pos_old + (1 - mid) * pos;
			dir = (slope_index * (pos1 - pos2) + dist * normal) / (glm::length(slope_index * (pos1 - pos2) + dist * normal) + gap);
			if (sim_time > cut_time)
			{
				pos = pos2 + reflect_index * glm::length(vel) * dir * dt;
				pos_old = pos2;
			}
			else
			{
				pos = pos + dist * normal;
				pos_old = pos;
			}

			collision_force[idx] = normal;
		}
		else
			collision_force[idx] = glm::vec3(0.0);
	}
	else
		collision_force[idx] = glm::vec3(0.0);
}

__device__ glm::vec3 compute_spring_force(int index, glm::vec3* g_pos_in, glm::vec3* g_pos_old_in,
	unsigned int* CSR_R, s_spring* CSR_C_SPRING,
	glm::vec3 pos, glm::vec3 vel, float k_spring)
{
	glm::vec3 force(0.0);
	int first_neigh = CSR_R[index];
	int time = 0;
	for (int k = first_neigh; k < CSR_R[index + 1]; k++)
	{
		float ks = k_spring;
		float kd = -0.5;

		int index_neigh = CSR_C_SPRING[k].end;
		volatile auto pos_neighData = g_pos_in[index_neigh];
		volatile auto pos_lastData = g_pos_old_in[index_neigh];
		glm::vec3 p2 = glm::vec3(pos_neighData.x, pos_neighData.y, pos_neighData.z);
		glm::vec3 p2_last = glm::vec3(pos_lastData.x, pos_lastData.y, pos_lastData.z);

		glm::vec3 v2 = (p2 - p2_last) / dt;
		glm::vec3 deltaP = pos - p2;

		deltaP += glm::vec3(FLT_EPSILON);    //avoid 0

		glm::vec3 deltaV = vel - v2;
		float dist = glm::length(deltaP);


		float original_length = CSR_C_SPRING[k].original;
		float leftTerm = -ks * (dist - original_length);
		float  rightTerm = kd * (glm::dot(deltaV, deltaP) / dist);
		glm::vec3 springForce = (leftTerm + rightTerm) * glm::normalize(deltaP);

		force += springForce;
	}
	return force;
}


__global__ void compute_face_normal(glm::vec3* g_pos_in, unsigned int* cloth_index, const unsigned int cloth_index_size, glm::vec3* cloth_face)
{
	unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int max_thread = cloth_index_size / 3;
	if (index >= max_thread)
		return;

	unsigned int f_index[3];
	for (int i = 0; i < 3; i++)
		f_index[i] = index * 3 + i;

	glm::vec3 vertex[3];
	for (int i = 0; i < 3; i++)
		vertex[i] = g_pos_in[cloth_index[f_index[i]]];  //find the fucking bug!

	glm::vec3 pos[3];
	for (int i = 0; i < 3; i++)
		pos[i] = glm::vec3(vertex[i].x, vertex[i].y, vertex[i].z);

	glm::vec3 side1, side2, normal;
	side1 = pos[1] - pos[0];
	side2 = pos[2] - pos[0];
	normal = glm::normalize(glm::cross(side1, side2));

	cloth_face[index] = normal;
}


__global__ void update_vbo_pos(glm::vec4* pos_vbo, glm::vec3* pos_cur, const unsigned int NUM_VERTICES)
{
	unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= NUM_VERTICES)
		return;

	auto pos = pos_cur[index];
	pos_vbo[index] = glm::vec4(pos.x, pos.y, pos.z, 1.0);
}

__global__ void verlet(glm::vec3* g_pos_in, glm::vec3* g_pos_old_in, glm::vec3* g_pos_out, glm::vec3* g_pos_old_out,
	unsigned int* CSR_R_STR, s_spring* CSR_C_STR, unsigned int* CSR_R_BD, s_spring* CSR_C_BD,
	D_BVH bvh, glm::vec3* collision_force,
	const unsigned int NUM_VERTICES, float* gpu_time, float sim_time, glm::vec3* detect_force)
{
	unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= NUM_VERTICES)
		return;

	volatile glm::vec3 posData = g_pos_in[index];
	volatile glm::vec3 posOldData = g_pos_old_in[index];


	glm::vec3 pos = glm::vec3(posData.x, posData.y, posData.z);
	glm::vec3 pos_old = glm::vec3(posOldData.x, posOldData.y, posOldData.z);
	glm::vec3 vel = (pos - pos_old) / dt;

	glm::vec3 gravity(gravit_x, gravit_y, gravit_z);
	glm::vec3 force = gravity * mass + vel * damp;
	glm::vec3 F_lift;
	glm::vec3 F_drag;
	force += compute_spring_force(index, g_pos_in, g_pos_old_in, CSR_R_STR, CSR_C_STR, pos, vel, spring_structure); // Compute structure spring force
	force += compute_spring_force(index, g_pos_in, g_pos_old_in, CSR_R_BD, CSR_C_BD, pos, vel, spring_bend); // Compute bend spring force
	// wind_begin
	F_lift = compute_wind_force_lift(index, g_pos_in, CSR_R_STR, CSR_C_STR, vel, pos, sim_time);
	F_drag = compute_wind_force_drag(index, g_pos_in, CSR_R_STR, CSR_C_STR, vel, pos, sim_time);
	glm::vec3 vec_wind = wind_velocity(F_lift, F_drag, sim_time);
	F_lift = compute_wind_force2_lift(index, g_pos_in, CSR_R_STR, CSR_C_STR, vel, pos, vec_wind);
	F_drag = compute_wind_force2_drag(index, g_pos_in, CSR_R_STR, CSR_C_STR, vel, pos, vec_wind);
	force = force + F_lift + F_drag;
	// wind_end
	force += compute_selfcollide_force(index, g_pos_in, NUM_VERTICES, CSR_R_STR, CSR_C_STR, CSR_R_BD, CSR_C_BD);
	glm::vec3 inelastic_force = glm::dot(collision_force[index], force) * collision_force[index];       //collision response force, if intersected, keep tangential
	force -= inelastic_force;
	glm::vec3 acc = force / mass;
	glm::vec3 tmp = pos;
	pos = pos + pos - pos_old + acc * dt * dt;
	pos_old = tmp;
	collision_response_projection(bvh, force, pos, pos_old, index, collision_force, vel, sim_time);
	//if (sim_time>100)
	//	collision_response_projection(cloth_bvh, force, pos, pos_old, index, collision_force);

	g_pos_out[index] = pos;
	g_pos_old_out[index] = pos_old;
	*gpu_time = sim_time;
	*detect_force = F_lift;

}
__global__ void compute_vbo_normal(glm::vec3* normals, unsigned int* CSR_R, unsigned int* CSR_C_adjface_to_vertex, glm::vec3* face_normal, const unsigned int NUM_VERTICES)
{

	unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= NUM_VERTICES)
		return;

	//compute point normal
	glm::vec3 normal(0.0);
	int first_face_index = CSR_R[index];
	for (int i = first_face_index; i < CSR_R[index + 1]; i++)
	{
		int findex = CSR_C_adjface_to_vertex[i];
		glm::vec3 fnormal = face_normal[findex];
		normal += fnormal;
	}
	normal = glm::normalize(normal);

	normals[index] = normal;
}
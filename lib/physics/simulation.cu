#include "hip/hip_runtime.h"
#include "simulator.h"

#include <iostream>
#include <fstream>
#include "watch.h"

using namespace std;
vector<glm::vec3> test(10000);
__global__ void compute_face_normal(glm::vec3* g_pos_in, unsigned int* cloth_index, const unsigned int cloth_index_size, glm::vec3* cloth_face);   //update cloth face normal
__global__ void verlet(glm::vec3 * g_pos_in, glm::vec3 * g_pos_old_in, glm::vec3 * g_pos_out, glm::vec3 * g_pos_old_out,
						unsigned int* CSR_R_STR, s_spring* CSR_C_STR, unsigned int* CSR_R_BD, s_spring* CSR_C_BD,
						D_BVH bvh, glm::vec3* d_collision_force,
						const unsigned int NUM_VERTICES);  //verlet intergration
__global__ void update_vbo_pos(glm::vec4* pos_vbo, glm::vec3* pos_cur, const unsigned int NUM_VERTICES);
__global__ void compute_vbo_normal(glm::vec3* normals, unsigned int* CSR_R, unsigned int* CSR_C_adjface_to_vertex, glm::vec3* face_normal, const unsigned int NUM_VERTICES);

Simulator::Simulator()
{
	
}

Simulator::~Simulator()
{
	hipFree(x_cur[0]);
	hipFree(x_cur[1]);
	hipFree(x_last[0]);
	hipFree(x_last[1]);
	hipFree(d_collision_force);
	hipFree(d_CSR_R);
	hipFree(d_CSR_C_adjface_to_vertex);
	hipFree(d_face_normals);

	hipFree(CSR_R_structure);
	hipFree(CSR_R_bend);
	hipFree(CSR_C_structure);
	hipFree(CSR_C_bend);

	delete cuda_bvh;
}

Simulator::Simulator(Mesh& sim_cloth, Mesh& body) :readID(0), writeID(1)
{
	init_cloth(sim_cloth);
	init_spring(sim_cloth);
	build_bvh(body);
}

void Simulator::init_cloth(Mesh& sim_cloth)
{
	//set heap size, the default is 8M
	size_t heap_size = 256 * 1024 * 1024;  
	hipDeviceSetLimit(hipLimitMallocHeapSize, heap_size);

	// Send the cloth's vertices to GPU
	const unsigned int vertices_bytes = sizeof(glm::vec3) * sim_cloth.vertices.size();
	safe_cuda(hipMalloc((void**)&x_cur[0], vertices_bytes));			 // cloth vertices
	safe_cuda(hipMalloc((void**)&x_cur[1], vertices_bytes));			 // cloth vertices
	safe_cuda(hipMalloc((void**)&x_last[0], vertices_bytes));	 // cloth old vertices
	safe_cuda(hipMalloc((void**)&x_last[1], vertices_bytes));	 // cloth old vertices
	safe_cuda(hipMalloc((void**)&d_collision_force, sizeof(glm::vec3) * sim_cloth.vertices.size()));  //collision response force
	safe_cuda(hipMemset(d_collision_force, 0, sizeof(glm::vec3) * sim_cloth.vertices.size()));    //initilize to 0

	x_cur_in = x_cur[readID];
	x_cur_out = x_cur[writeID];
	x_last_in = x_last[readID];
	x_last_out = x_last[writeID];

	vector<glm::vec3> tem_vertices(sim_cloth.vertices.size());
	for (int i=0;i< sim_cloth.vertices.size();i++)
	{
		tem_vertices[i] = glm::vec3(sim_cloth.vertices[i]);   // glm::vec4 -> glm::vec3
	}

	safe_cuda(hipMemcpy(x_cur[0], &tem_vertices[0], vertices_bytes, hipMemcpyHostToDevice));
	safe_cuda(hipMemcpy(x_last[0], &tem_vertices[0], vertices_bytes, hipMemcpyHostToDevice));


	//hipMemcpy(&test[0], x_cur[0], sizeof(glm::vec3) * sim_cloth.vertices.size(), hipMemcpyDeviceToHost);
	//hipMemcpy(&test, x_cur[1], sizeof(glm::vec3), hipMemcpyDeviceToHost);
	//hipMemcpy(&test, d_collision_force, sizeof(glm::vec3)+1, hipMemcpyDeviceToHost);
	//����normal��������ݣ�ÿ�����ڽӵ�������� + ÿ�����3���������
	vector<unsigned int> TEM_CSR_R;
	vector<unsigned int> TEM_CSR_C_adjface;
	get_vertex_adjface(sim_cloth, TEM_CSR_R, TEM_CSR_C_adjface);

	safe_cuda(hipMalloc((void**)&d_CSR_R, sizeof(unsigned int) * TEM_CSR_R.size()));
	safe_cuda(hipMalloc((void**)&d_CSR_C_adjface_to_vertex, sizeof(unsigned int) * TEM_CSR_C_adjface.size()));
	safe_cuda(hipMemcpy(d_CSR_R, &TEM_CSR_R[0], sizeof(unsigned int) * TEM_CSR_R.size(), hipMemcpyHostToDevice));
	safe_cuda(hipMemcpy(d_CSR_C_adjface_to_vertex, &TEM_CSR_C_adjface[0], sizeof(unsigned int) * TEM_CSR_C_adjface.size(), hipMemcpyHostToDevice));
	
	safe_cuda(hipMalloc((void**)&d_face_normals, sizeof(glm::vec3) * sim_cloth.faces.size()));    //face normal
}

void Simulator::init_spring(Mesh& sim_cloth)
{
	cout << "build springs" << endl;
	// Construct structure and bend springs in GPU
	Springs springs(&sim_cloth);
	
	vector<unsigned int> TEM_CSR_R_structure, TEM_CSR_R_bend;
	vector<s_spring> TEM_CSR_C_structure, TEM_CSR_C_bend;

	springs.CSR_structure_spring(&sim_cloth, TEM_CSR_R_structure, TEM_CSR_C_structure);
	springs.CSR_bend_spring(&sim_cloth, TEM_CSR_R_bend, TEM_CSR_C_bend);

	safe_cuda(hipMalloc((void**)&CSR_R_structure, TEM_CSR_R_structure.size() * sizeof(unsigned int)));
	safe_cuda(hipMalloc((void**)&CSR_R_bend, TEM_CSR_R_bend.size() * sizeof(unsigned int)));
	safe_cuda(hipMalloc((void**)&CSR_C_structure, TEM_CSR_C_structure.size() * sizeof(s_spring)));
	safe_cuda(hipMalloc((void**)&CSR_C_bend, TEM_CSR_C_bend.size() * sizeof(s_spring)));

	safe_cuda(hipMemcpy(CSR_R_structure, &TEM_CSR_R_structure[0], TEM_CSR_R_structure.size() * sizeof(unsigned int), hipMemcpyHostToDevice));
	safe_cuda(hipMemcpy(CSR_R_bend, &TEM_CSR_R_bend[0], TEM_CSR_R_bend.size() * sizeof(unsigned int), hipMemcpyHostToDevice));
	safe_cuda(hipMemcpy(CSR_C_structure, &TEM_CSR_C_structure[0], TEM_CSR_C_structure.size() * sizeof(s_spring), hipMemcpyHostToDevice));
	safe_cuda(hipMemcpy(CSR_C_bend, &TEM_CSR_C_bend[0], TEM_CSR_C_bend.size() * sizeof(s_spring), hipMemcpyHostToDevice));
	
	cout << "springs build successfully!" << endl;
}

void Simulator::build_bvh(Mesh& body)
{
	stop_watch watch;
	watch.start();
	Mesh bvh_body = body;   // for bvh consttruction
	bvh_body.vertex_extend(0.007);

	watch.start();
	cuda_bvh = new BVHAccel(bvh_body);
	watch.stop();
	cout << "bvh build done free time elapsed: " << watch.elapsed() << "us" << endl;
}


void Simulator::simulate(Mesh* sim_cloth)
{
	//cuda kernel compute .........
	/*
	for (int i = 0; i<1000; i++)
	{
		cuda_verlet(sim_cloth->vertices.size());
		//cuda_update_vbo(sim_cloth);     // update array buffer for opengl
		hipMemcpy(&test[0], x_cur_out, sizeof(glm::vec3) * sim_cloth->vertices.size(), hipMemcpyDeviceToHost);
		swap_buffer();
	}
	*/
	cuda_verlet(sim_cloth->vertices.size());
	//	cuda_update_vbo(sim_cloth);     // update array buffer for opengl
	//test.resize(sim_cloth->vertices.size());
	hipMemcpy(&sim_cloth->onestep_vertices[0], x_cur_out, sizeof(glm::vec3) * sim_cloth->vertices.size(), hipMemcpyDeviceToHost);
	swap_buffer();
}

void Simulator::get_vertex_adjface(Mesh& sim_cloth, vector<unsigned int>& CSR_R, vector<unsigned int>& CSR_C_adjface)
{
	vector<vector<unsigned int>> adjaceny(sim_cloth.vertices.size());
	for(int i=0;i<sim_cloth.faces.size();i++)
	{
		unsigned int f[3];
		for(int j=0;j<3;j++)
		{
			f[j] = sim_cloth.faces[i].vertex_index[j];
			adjaceny[f[j]].push_back(i);
		}
	}

	// i-th vertex adjacent face start_index = CSR_R[i], end_index = CSR_R[i+1]
	// then you can acess CSR_C_adjface[start_index->end_index]
	unsigned int start_idx = 0;
	for(int i=0;i<adjaceny.size();i++)
	{
		CSR_R.push_back(start_idx);
		start_idx += adjaceny[i].size();

		for(int j=0;j<adjaceny[i].size();j++)
		{
			CSR_C_adjface.push_back(adjaceny[i][j]);
		}
	}

	CSR_R.push_back(start_idx);
}

void Simulator::cuda_verlet(const unsigned int numParticles)
{
	unsigned int numThreads, numBlocks;
	
	computeGridSize(numParticles, 512, numBlocks, numThreads);
	verlet <<< numBlocks, numThreads >>>(x_cur_in,x_last_in, x_cur_out, x_last_out,
										CSR_R_structure, CSR_C_structure, CSR_R_bend, CSR_C_bend,
										*cuda_bvh->d_bvh, d_collision_force,
										numParticles);

	// stop the CPU until the kernel has been executed
	safe_cuda(hipDeviceSynchronize());
}

void Simulator::computeGridSize(unsigned int n, unsigned int blockSize, unsigned int &numBlocks, unsigned int &numThreads)
{
	numThreads = min(blockSize, n);
	numBlocks = (n % numThreads != 0) ? (n / numThreads + 1) : (n / numThreads);
}

void Simulator::swap_buffer()
{
	swap(readID, writeID);

	x_cur_in = x_cur[readID];
	x_cur_out = x_cur[writeID];
	x_last_in = x_last[readID];
	x_last_out = x_last[writeID];
}

void Simulator::update_vertex(glm::vec3 new_value, const unsigned int idx)
{
	safe_cuda(hipMemcpy(&x_cur_in[idx], &new_value[0], sizeof(glm::vec3), hipMemcpyHostToDevice));
	safe_cuda(hipMemcpy(&x_last_in[idx], &new_value[0], sizeof(glm::vec3), hipMemcpyHostToDevice));
}

